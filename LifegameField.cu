#include "hip/hip_runtime.h"
#include "LifegameField.h"
#include <random>
#include <iostream>
#include<hip/hip_runtime.h>
#include<opencv2/cudev.hpp>

__global__ void CorrectFieldKernel(cv::cudev::GlobPtrSz<uchar> field, cv::cudev::GlobPtrSz<uchar> buf);

__global__ void AdvanceGenKernel(cv::cudev::GlobPtrSz<uchar> field, cv::cudev::GlobPtrSz<uchar> buf, unsigned int gen);


LifegameField::LifegameField(int row, int col)
{
  cv::Mat tmp(cv::Size(col+2, row+2), CV_8UC1);

  hipMallocManaged(&managedField, tmp.rows*tmp.cols*tmp.channels());
  hipMallocManaged(&managedBuf, tmp.rows*tmp.cols*tmp.channels());

  int w = tmp.cols;
  int h = tmp.rows;

  cpuField = cv::Mat(cv::Size(w,h), CV_8UC1, managedField);
  gpuField = cv::cuda::GpuMat(cv::Size(w,h), CV_8UC1, managedField);

  cpuBuf = cv::Mat(cv::Size(w,h), CV_8UC1, managedBuf);
  gpuBuf = cv::cuda::GpuMat(cv::Size(w,h), CV_8UC1, managedBuf);

  memcpy(managedField, tmp.data, w*h);
  memcpy(managedBuf, tmp.data, w*h);

  RandomInit();
  CorrectField();
}

LifegameField::~LifegameField()
{
  hipFree(managedBuf);
  hipFree(managedField);
}

unsigned int LifegameField::AdvanceGen()
{
  gen ++;
  //  RandomInit();
  
  cv::cudev::GlobPtrSz<uchar> pField = 
    cv::cudev::globPtr(gpuField.ptr(0), gpuField.step, gpuField.rows, gpuField.cols * gpuField.channels());
  cv::cudev::GlobPtrSz<uchar> pBuf = 
    cv::cudev::globPtr(gpuBuf.ptr(0), gpuBuf.step, gpuBuf.rows, gpuBuf.cols * gpuBuf.channels()); 
  const dim3 block(32, 8);
  const dim3 grid(cv::cudev::divUp(gpuField.cols, block.x), cv::cudev::divUp(gpuField.rows, block.y));
  AdvanceGenKernel<<<grid, block>>>(pField, pBuf, gen);
  hipDeviceSynchronize();
  memcpy(managedField, managedBuf, gpuField.rows* gpuField.cols);   
  CorrectField();
  return gen;
}


void LifegameField::CorrectField()
{
  cv::cudev::GlobPtrSz<uchar> pField = 
    cv::cudev::globPtr(gpuField.ptr(0), gpuField.step, gpuField.rows, gpuField.cols * gpuField.channels());
  cv::cudev::GlobPtrSz<uchar> pBuf = 
    cv::cudev::globPtr(gpuBuf.ptr(0), gpuBuf.step, gpuBuf.rows, gpuBuf.cols * gpuBuf.channels()); 
  const dim3 block(32, 8);
  const dim3 grid(cv::cudev::divUp(gpuField.cols, block.x), cv::cudev::divUp(gpuField.rows, block.y));
  CorrectFieldKernel<<<grid, block>>>(pField, pBuf);
  hipDeviceSynchronize();
  memcpy(managedField, managedBuf, gpuField.rows* gpuField.cols);
}

void LifegameField::RandomInit()
{
  std::mt19937 mt{std::random_device{}()};
  std::uniform_int_distribution<unsigned int> dist(0,2);

  for(int c = 1; c < cpuField.cols-1; c ++){
    for(int r = 1; r < cpuField.rows-1; r ++){
      unsigned int v = dist(mt);
      if(v == 0){
	cpuField.at<unsigned char>(r,c) = 1;
      }
      else{
	cpuField.at<unsigned char>(r,c) = 0;
      }
    }
  }
}



__global__ void CorrectFieldKernel(cv::cudev::GlobPtrSz<uchar> field, cv::cudev::GlobPtrSz<uchar> buf)
{
  const int c = blockDim.x * blockIdx.x + threadIdx.x;
  const int r = blockDim.y * blockIdx.y + threadIdx.y;

  int cmax = field.cols - 1;
  int rmax = field.rows - 1;

  if(c >= field.cols || r >= field.rows){
    return;
  }

  if(c == 0 && r == 0){
    buf.data[r * field.step + c] = field.data[(rmax-1) * field.step + (cmax-1)];
    return;
  }
  
  if(c == 0 && r == rmax){
    buf.data[r * field.step + c] = field.data[(1) * field.step + (cmax-1)];
    return;
  }

  if(c == cmax && r == 0){
    buf.data[r * field.step + c] = field.data[(rmax-1) * field.step + (1)];
    return;
  }
  
  if(c == cmax && r == 0){
    buf.data[r * field.step + c] = field.data[(1) * field.step + (1)];
    return;
  }

  if(c == 0){
    buf.data[r * field.step + c] = field.data[(r) * field.step + (cmax-1)];
    return;
  }

  if(c == cmax){
    buf.data[r * field.step + c] = field.data[(r) * field.step + (1)];
    return;
  }

  if(r == 0){
    buf.data[r * field.step + c] = field.data[(rmax-1) * field.step + (c)];
    return;
  }

  if(r == rmax){
    buf.data[r * field.step + c] = field.data[(1) * field.step + (c)];
    return;
  }
  
  buf.data[r * field.step + c] = field.data[r * field.step + c]; 

}

__global__ void AdvanceGenKernel(cv::cudev::GlobPtrSz<uchar> field, cv::cudev::GlobPtrSz<uchar> buf, unsigned


				 int gen)
{
  const int c = blockDim.x * blockIdx.x + threadIdx.x;
  const int r = blockDim.y * blockIdx.y + threadIdx.y;

  int cmax = field.cols - 1;
  int rmax = field.rows - 1;

  if(c <= 0 || c >= cmax || r <= 0 || r >= rmax){
    return;
  }

  int center = field.data[(r) * field.step + (c)];

  int neighbor = 0;
  neighbor += field.data[(r-1) * field.step + (c-1)];
  neighbor += field.data[(r-1) * field.step + (c)];
  neighbor += field.data[(r-1) * field.step + (c+1)];
  neighbor += field.data[(r) * field.step + (c-1)];
  neighbor += field.data[(r) * field.step + (c+1)];
  neighbor += field.data[(r+1) * field.step + (c-1)];
  neighbor += field.data[(r+1) * field.step + (c)];
  neighbor += field.data[(r+1) * field.step + (c+1)];

  //birth
  if(center == 0 && ( neighbor == 3 || neighbor == 6)){
    buf.data[(r) * field.step + (c)] = 1;
    return;
  }

  if(center == 0){
    buf.data[(r) * field.step + (c)] = 0;
    return;
  }

  //center == 1
  //survival
  if(neighbor == 2 || neighbor == 3){
    buf.data[(r) * field.step + (c)] = 1;
    return;
  }
  
  //underpopulation
  //overpopulation
  buf.data[(r) * field.step + (c)] = 0;
  return;
}
